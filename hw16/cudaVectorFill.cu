#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorFillKernel(int N, double *x){

  const int thread = threadIdx.x;
  const int block = blockIdx.x;
  const int dim = blockDim.x;
  const int grid = gridDim.x;

  int n = thread + block*dim; // n in range [0,grid*dim)

  while(n<N){
    x[n] = 999;
    n += grid*dim;
  }
}

// will execute on the HOST (CPU)
int main(int argc, char **argv){
    
  int G = 4; // 10 thread blocks
  int B = 64; // 20 threads per thread-block

  int N = 10000;
  double *c_x;
  double *h_x = (double*) calloc(N, sizeof(double));

  hipMalloc(&c_x, N*sizeof(double));

  vectorFillKernel <<< G, B >>> (N, c_x);

  hipMemcpy(h_x, c_x, N*sizeof(double), hipMemcpyDeviceToHost);
  h_x[0]=0;  

  for(int n=0;n<N;++n){
    if(n == 0){
      h_x[0]=0;
      printf("h_x[%d] = %f\n", n, h_x[n]);
    }
    else {
      h_x[n] = h_x[n-1]+ 1;
      printf("h_x[%d] = %f\n", n, h_x[n]);
    }
  }

  hipFree(c_x);
}
