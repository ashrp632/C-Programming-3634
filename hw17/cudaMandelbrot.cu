#include "hip/hip_runtime.h"
/*

To compile:
make cudaMandelbrot

To run:
./cudaMandelbrot

*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

extern "C"
{
#include "png_util.h"
}

__global__ void mandelbrotKernel(const int Nx, 
	   			       	    const int Ny, 
					    	      	   const float xmin,
									 const float ymin,
											 const float dx, 
											       	     	  const float dy,
															 float * count){
  
  int n = threadIdx.x + blockDim.x*blockIdx.x;
  int m = threadIdx.y + blockDim.y*blockIdx.y;
  
  const float cx = xmin + n*dx;
  const float cy = ymin + m*dy;
  
  float x = 0;
  float y = 0;
  float xold = 0;
  float yold = 0;
  int Nit = 200;
  int t, cnt=0;
  for(t=0;t<Nit;++t){
    
    // x,y are updated to
    // (x^2 - y^2 + cx, 2*x*y+cre)
    
    // HW17 TASKS:
    // 1. change the update formula using an alternative function in x,y,xold,yold
    //   [ some ideas here: https://math.stackexchange.com/questions/1099/mandelbrot-like-sets-for-functions-other-than-fz-z2c ]
    // 2. zoom in on an interesting feature
    // 3. run the code 
    // 4. use scp to copy the generated png to your laptop
    // 5. upload the png to canvas

    {
      xold = x;
      yold = y;

      float xnew = cx;
      xnew += x*x;
      xnew -= y*y; 
      
      float ynew = cy;
      ynew += 2.*x*y;
      
      x = xnew;
      y = ynew;
    }

    // check if iterations escaped the radius 2 circle 
    float r = x*x;
    r +=y*y;
    
    if(r<4.f)
      ++cnt;
  }
  
  count[n + m*Nx] = cnt;
  
}


int main(int argc, char **argv){

  int Nx = 4096;
  int Ny = 4096;

  /* box containing sample points */
  float xcent = -.759856;
  float ycent= .125547;
  float diam  = 0.151579;

  /* box is: [xmin,xmax] x [ymin,ymax] */
  float xmin = xcent -0.5*diam;
  float xmax = xcent +0.5*diam;
  float ymin = ycent -0.5*diam;
  float ymax = ycent +0.5*diam;

  float dx = (xmax-xmin)/(Nx-1.f);
  float dy = (ymax-ymin)/(Ny-1.f);
  
  float *h_count = (float*) calloc(Nx*Ny, sizeof(float));
  float *c_count;

  hipMalloc(&c_count, Nx*Ny*sizeof(float));

  // call mandelbrot from here
  // we will use a thread-block of NTxNT threads
  // every thread will only iterate one point
  int NT = 8;
  dim3 B(NT,NT);
  dim3 G((Nx+NT-1)/NT, (Ny+NT-1)/NT);
  
  hipEvent_t tic, toc;
  hipEventCreate(&tic);
  hipEventCreate(&toc);

  hipEventRecord(tic);

  mandelbrotKernel <<<G,B>>> (Nx, Ny, xmin, ymin, dx, dy, c_count);

  hipEventRecord(toc);

  hipDeviceSynchronize();
  float elapsed;
  hipEventElapsedTime(&elapsed, tic, toc);
  elapsed/=1000;
  printf("elapsed time %g\n", elapsed);

  // copy data from DEVICE to HOST
  hipMemcpy(h_count, c_count, Nx*Ny*sizeof(float), hipMemcpyDeviceToHost);

  // output data from HOST
  FILE *png = fopen("cudaMandelbrot.png", "w");
  write_hot_png(png, Nx, Ny, h_count, 0, 80);
  fclose(png);

}
